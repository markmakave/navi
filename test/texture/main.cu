#include "hip/hip_runtime.h"
#include "cuda/matrix.cuh"
#include "cuda/cuda.hpp"
#include "base/color.hpp"

#include <hip/hip_runtime.h>

__global__ void kernel(hipTextureObject_t texture) {
    for (int i = -3; i < 6; ++i)
    {
        for (int j = -3; j < 6; ++j)
            printf("%d ", tex2D<lm::gray>(texture, j, i));
        printf("\n");
    }  
}

int main()
{
    lm::cuda::matrix<lm::gray> m(3, 3);
    for (int i = 0; i < 9; ++i)
        m(i / 3, i % 3) = i;

    hipArray_t array;
    auto channed_desc = hipCreateChannelDesc<lm::gray>();
    hipMallocArray(&array, &channed_desc, m.width(), m.height());

    hipMemcpyToArray(array, 0, 0, m.data(), m.size() * sizeof(lm::gray), hipMemcpyDeviceToDevice);

    hipTextureDesc tex_desc = {};
    tex_desc.normalizedCoords = false;
    tex_desc.filterMode = hipFilterModePoint;
    tex_desc.addressMode[0] = hipAddressModeBorder;
    tex_desc.addressMode[1] = hipAddressModeBorder;
    tex_desc.readMode = hipReadModeElementType;

    hipResourceDesc res_desc = {};
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = array;

    hipTextureObject_t texture;
    hipCreateTextureObject(&texture, &res_desc, &tex_desc, nullptr);

    kernel <<<1, 1>>> (texture);
    hipDeviceSynchronize();

    return 0;
}
