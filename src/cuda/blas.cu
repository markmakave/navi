#include "hip/hip_runtime.h"
#include "cuda/blas.cuh"

template <>
__global__
void
lm::cuda::blas::add_kernel(
    const lm::cuda::array<float> x,
    const lm::cuda::array<float> y,
          lm::cuda::array<float> z
) {
    i64 i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= x.size()) return;
    z(i) = x(i) + y(i);
}

template <>
__global__
void
lm::cuda::blas::sub_kernel(
    const lm::cuda::array<float> x,
    const lm::cuda::array<float> y,
          lm::cuda::array<float> z
) {
    i64 i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= x.size()) return;
    z(i) = x(i) - y(i);
}

template <>
__global__
void
lm::cuda::blas::mul_kernel(
    const lm::cuda::array<float> x,
    const lm::cuda::array<float> y,
          lm::cuda::array<float> z
) {
    i64 i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= x.size()) return;
    z(i) = x(i) * y(i);
}

__device__
float
lm::cuda::blas::sigmoid(float x)
{
    return x > 0 ? x : 0;
}

__device__
float
lm::cuda::blas::sigmoid_derivative(float x)
{
    return x > 0 ? 1 : 0;
}

template <>
__global__
void
lm::cuda::blas::sigmoid_kernel(
    const array<float> x,
          array<float> y
) {
    i64 i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= x.size()) return;
    y(i) = sigmoid(x(i));
}

template <>
__global__
void
lm::cuda::blas::sigmoid_derivative_kernel(
    const array<float> x,
          array<float> y
) {
    i64 i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= x.size()) return;
    y(i) = sigmoid_derivative(x(i));
}
